#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <random>
#include <limits>
#include <cassert>
#include <algorithm>
#include <functional>
#include <string>
#include <array>
#include <tbb/parallel_for.h>
#include <tbb/parallel_invoke.h>
#include <tbb/parallel_sort.h>

extern "C" {
#include <sys/time.h>
}

using std::cout;
using std::endl;
using std::vector;
using std::function;
using std::string;
using std::swap;

// PARAMS
static const long MAXSIZE = pow(2,33) / sizeof(int); // 4GB (can't do 8, since mergesort needs copies of the list)
//static const long MAXSIZE = pow(2,24) / sizeof(int); // 4GB (can't do 8, since mergesort needs copies of the list)a
static const long STARTSIZE = pow(2, 20) / sizeof(int); // 1MB
static const int CUTOFFQS = 500;
static const int CUTOFFMSSORT = 500;
static const int CUTOFFMSMERGE = 2000;
static const bool INTROSPECTIVE = true;

// STRUCTS
struct SortFunc {
  function<void(vector<int>::iterator, vector<int>::iterator)> func;
  string name;
  vector<double> times;

  SortFunc(function<void(vector<int>::iterator, vector<int>::iterator)> f, string n) {
    func = f;
    name = n;
  }
};


// UTIL
string readableSize(long size) {
  int i = 0;
  const char* units[] = {"B", "kB", "MB", "GB", "TB", "PB", "EB", "ZB", "YB"};
  while (size > 1024) {
    size /= 1024;
    i++;
  }
  return std::to_string(size) + " " + units[i];
}

double getWallTime() {
  timeval timeofday;
  gettimeofday( &timeofday, NULL );
  return timeofday.tv_sec + timeofday.tv_usec / 1000000.0;
}

vector<int> createRandomList(long n) {
  assert(n>=0);
  auto gen = std::default_random_engine();
  auto dist =  std::uniform_int_distribution<int>(std::numeric_limits<int>::min(), std::numeric_limits<int>::max());
  vector<int> randomList(n);
  std::generate(randomList.begin(), randomList.end(), [=]()mutable{ return dist(gen);});
  assert(randomList.size() == (unsigned long) n);
  return randomList; //Optimizer will do named return type move
}

void parallelGnuSort(vector<int>::iterator l, vector<int>::iterator r) {
  __gnu_parallel::sort(l, r);
}

void parallelCudaSort(vector<int>::iterator l, vector<int>::iterator r) {
  
}


double runTest(SortFunc &sortFunc, long i){
  assert(i >= 0);
  auto listToSort = createRandomList(i);
  auto startTime = getWallTime();
  cout << " [-] SORT: " << sortFunc.name << "\t " << std::flush;
  sortFunc.func(listToSort.begin(), listToSort.end());
  auto totalTime = getWallTime() - startTime;
  //assert(is_sorted(listToSort.begin(), listToSort.end()));
  printf("TIME: %.3f seconds\n", totalTime);
  return totalTime;
}

int main(int argc, char *argv[]) {
  std::array<SortFunc, 7> sortFunctions = {
    SortFunc{::parallelGnuSort, "Parallel GNU Sort"},
    SortFunc{::parallelCudaSort, "Parallel CUDA Sort"}
  };

  for (long i = STARTSIZE; i < MAXSIZE; i*=2){
    cout << "Running tests on size " << readableSize(i*sizeof(int)) << endl;
    for (auto& sortFunc : sortFunctions) {
      sortFunc.times.push_back(runTest(sortFunc, i));
    }
  }
  return 0;
}
